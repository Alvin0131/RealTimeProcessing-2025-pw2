#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include <iostream>
#include <limits>

using namespace cv;
using namespace std;

#define BLOCK_DIM 16 // Block size (16x16 threads)
#define MAX_KERNEL_SIZE 13 // Maximum kernel size

// CUDA kernel for Gaussian blur
__global__ void gaussianBlurKernelShared(uchar3* input, uchar3* output, int width, int height, double* kernel, int kernelSize) {
    __shared__ uchar3 sharedBlock[BLOCK_DIM + MAX_KERNEL_SIZE][BLOCK_DIM + MAX_KERNEL_SIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int localX = threadIdx.x;
    int localY = threadIdx.y;

    int k = kernelSize / 2;
    int sharedX = localX + k;
    int sharedY = localY + k;

    // Load the image block into shared memory
    if (x < width && y < height) {
        sharedBlock[sharedY][sharedX] = input[y * width + x];
    } else {
        sharedBlock[sharedY][sharedX] = make_uchar3(0, 0, 0); // Handle out-of-bounds
    }

    // Load halo regions
    if (localX < k) {
        sharedBlock[sharedY][localX] = input[y * width + max(x - k, 0)];
        sharedBlock[sharedY][localX + blockDim.x + k] = input[y * width + min(x + blockDim.x, width - 1)];
    }
    if (localY < k) {
        sharedBlock[localY][sharedX] = input[max(y - k, 0) * width + x];
        sharedBlock[localY + blockDim.y + k][sharedX] = input[min(y + blockDim.y, height - 1) * width + x];
    }

    __syncthreads();

    // Apply Gaussian blur
    if (x < width && y < height) {
        double r = 0.0, g = 0.0, b = 0.0;

        for (int ky = -k; ky <= k; ky++) {
            for (int kx = -k; kx <= k; kx++) {
                uchar3 pixel = sharedBlock[sharedY + ky][sharedX + kx];
                double kval = kernel[(ky + k) * kernelSize + (kx + k)];
                r += pixel.x * kval;
                g += pixel.y * kval;
                b += pixel.z * kval;
            }
        }

        uchar3 out;
        out.x = static_cast<uchar>(r);
        out.y = static_cast<uchar>(g);
        out.z = static_cast<uchar>(b);
        output[y * width + x] = out;
    }
}


// CUDA function for Gaussian blur
extern "C"
void gaussianBlurCUDA(unsigned char* inputPtr, unsigned char* outputPtr, int width, int height, double* kernel, int kernelSize) {
    uchar3* d_input;
    uchar3* d_output;
    double* d_kernel;

    size_t imgSize = width * height * sizeof(uchar3);
    size_t kernelSizeBytes = kernelSize * kernelSize * sizeof(double);

    // Allocate memory on device
    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);
    hipMalloc(&d_kernel, kernelSizeBytes);

    // Copy data to device
    hipMemcpy(d_input, inputPtr, imgSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernelSizeBytes, hipMemcpyHostToDevice);

    float bestTime = std::numeric_limits<float>::max();
    int bestBx = 0, bestBy = 0;

    std::cout << "Benchmarking CUDA block sizes for Gaussian blur..." << std::endl;

    for (int bx = 4; bx <= 32; bx += 4) {
        for (int by = 4; by <= 32; by += 4) {
            dim3 threadsPerBlock(bx, by);
            dim3 numBlocks((width + bx - 1) / bx, (height + by - 1) / by);

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start);
            gaussianBlurKernelShared<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height, d_kernel, kernelSize);
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            std::cout << "Block (" << bx << "," << by << ") - Time: " << milliseconds << " ms" << std::endl;

            if (milliseconds < bestTime) {
                bestTime = milliseconds;
                bestBx = bx;
                bestBy = by;
            }

            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }

    // Final execution with best block size
    std::cout << "Best block size: (" << bestBx << "," << bestBy << ") - Time: " << bestTime << " ms" << std::endl;

    dim3 threadsPerBlock(bestBx, bestBy);
    dim3 numBlocks((width + bestBx - 1) / bestBx, (height + bestBy - 1) / bestBy);

    gaussianBlurKernelShared<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height, d_kernel, kernelSize);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(outputPtr, d_output, imgSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

// CUDA kernel for denoising (adaptive Gaussian blur)
__global__ void adaptiveDenoiseKernelShared(uchar3* input, uchar3* output, int width, int height, int neighborhoodSize, double factorRatio) {
    __shared__ uchar3 sharedBlock[BLOCK_DIM + MAX_KERNEL_SIZE][BLOCK_DIM + MAX_KERNEL_SIZE];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int localX = threadIdx.x;
    int localY = threadIdx.y;

    int pad = neighborhoodSize / 2;
    int sharedX = localX + pad;
    int sharedY = localY + pad;

    // Load the image block into shared memory
    if (x < width && y < height) {
        sharedBlock[sharedY][sharedX] = input[y * width + x];
    } else {
        sharedBlock[sharedY][sharedX] = make_uchar3(0, 0, 0); // Handle out-of-bounds
    }

    // Load halo regions
    if (localX < pad) {
        sharedBlock[sharedY][localX] = input[y * width + max(x - pad, 0)];
        sharedBlock[sharedY][localX + blockDim.x + pad] = input[y * width + min(x + blockDim.x, width - 1)];
    }
    if (localY < pad) {
        sharedBlock[localY][sharedX] = input[max(y - pad, 0) * width + x];
        sharedBlock[localY + blockDim.y + pad][sharedX] = input[min(y + blockDim.y, height - 1) * width + x];
    }

    __syncthreads();

    // Apply adaptive denoising
    if (x < width && y < height) {
        double r = 0.0, g = 0.0, b = 0.0;
        double weightSum = 0.0;

        for (int ky = -pad; ky <= pad; ky++) {
            for (int kx = -pad; kx <= pad; kx++) {
                uchar3 pixel = sharedBlock[sharedY + ky][sharedX + kx];
                double weight = exp(-(kx * kx + ky * ky) / (2 * factorRatio * factorRatio));
                r += pixel.x * weight;
                g += pixel.y * weight;
                b += pixel.z * weight;
                weightSum += weight;
            }
        }

        uchar3 out;
        out.x = static_cast<uchar>(r / weightSum);
        out.y = static_cast<uchar>(g / weightSum);
        out.z = static_cast<uchar>(b / weightSum);
        output[y * width + x] = out;
    }
}
// CUDA function for adaptive denoising
extern "C"
void adaptiveDenoiseCUDA(unsigned char* inputPtr, unsigned char* outputPtr, int width, int height, int neighborhoodSize, double factorRatio) {
    uchar3* d_input;
    uchar3* d_output;

    size_t imgSize = width * height * sizeof(uchar3);

    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);

    hipMemcpy(d_input, inputPtr, imgSize, hipMemcpyHostToDevice);

    float bestTime = std::numeric_limits<float>::max();
    int bestBx = 0, bestBy = 0;

    std::cout << "Benchmarking CUDA adaptive denoise with neighborhood size " << neighborhoodSize << "..." << std::endl;

    for (int bx = 4; bx <= 32; bx += 4) {
        for (int by = 4; by <= 32; by += 4) {
            dim3 threadsPerBlock(bx, by);
            dim3 numBlocks((width + bx - 1) / bx, (height + by - 1) / by);

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start);
            adaptiveDenoiseKernelShared<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height, neighborhoodSize, factorRatio);
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            std::cout << "Block (" << bx << "x" << by << ") -> Time: " << milliseconds << " ms" << std::endl;

            if (milliseconds < bestTime) {
                bestTime = milliseconds;
                bestBx = bx;
                bestBy = by;
            }

            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }

    std::cout << "Best block size for adaptive denoise: (" << bestBx << "x" << bestBy << ") with time: " << bestTime << " ms" << std::endl;

    // Run the best configuration and store the result
    dim3 bestThreads(bestBx, bestBy);
    dim3 bestBlocks((width + bestBx - 1) / bestBx, (height + bestBy - 1) / bestBy);
    adaptiveDenoiseKernelShared<<<bestBlocks, bestThreads>>>(d_input, d_output, width, height, neighborhoodSize, factorRatio);
    hipDeviceSynchronize();


    hipMemcpy(outputPtr, d_output, imgSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}